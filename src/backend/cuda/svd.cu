/*******************************************************
 * Copyright (c) 2014, ArrayFire
 * All rights reserved.
 *
 * This file is distributed under 3-clause BSD license.
 * The complete license agreement can be obtained at:
 * http://arrayfire.com/licenses/BSD-3-Clause
 ********************************************************/

#include <svd.hpp>
#include <err_common.hpp>

#if defined(WITH_CUDA_LINEAR_ALGEBRA)

#include <cusolverDnManager.hpp>
#include "transpose.hpp"
#include <memory.hpp>
#include <copy.hpp>
#include <math.hpp>
#include <err_common.hpp>

namespace cuda
{
    using cusolver::getDnHandle;

    template <typename T>
    struct gesvd_func_def_t {
        typedef hipsolverStatus_t (*gesvd_func_def)(hipsolverHandle_t, char, char, int,
                                                   int, T *, int, T *, T *, int, T *, int,
                                                   T *, int, T *, int *);
    };

    template<typename T>
    struct gesvd_buf_func_def_t {
        typedef hipsolverStatus_t (*gesvd_buf_func_def)(hipsolverHandle_t, int, int,
                                                       int *);
    };

#define SVD_FUNC_DEF(FUNC)                                                              \
    template <typename T>                                                               \
    typename FUNC##_func_def_t<T>::FUNC##_func_def FUNC##_func();                       \
                                                                                        \
    template<typename T>                                                                \
    typename FUNC##_buf_func_def_t<T>::FUNC##_buf_func_def                              \
    FUNC##_buf_func();

#define SVD_FUNC(FUNC, TYPE, PREFIX)                                                    \
    template <>                                                                         \
    typename FUNC##_func_def_t<TYPE>::FUNC##_func_def FUNC##_func<TYPE>()               \
    {                                                                                   \
        return (FUNC##_func_def_t<TYPE>::FUNC##_func_def) & cusolverDn##PREFIX##FUNC;   \
    }                                                                                   \
                                                                                        \
    template<> typename FUNC##_buf_func_def_t<TYPE>::FUNC##_buf_func_def                \
    FUNC##_buf_func<TYPE>()                                                             \
    {                                                                                   \
        return (FUNC##_buf_func_def_t<TYPE>::FUNC##_buf_func_def) &                     \
               cusolverDn##PREFIX##FUNC##_bufferSize;                                   \
    }

    SVD_FUNC_DEF(gesvd)
    SVD_FUNC(gesvd, float, S)
    SVD_FUNC(gesvd, double, D)
//SVD_FUNC(gesvd , cfloat , C)
//SVD_FUNC(gesvd , cdouble, Z)

    template <typename T>
    void svdInPlace(Array<T> &s, Array<T> &u, Array<T> &vt, Array<T> &in)
    {
        dim4 iDims = in.dims();
        int M = iDims[0];
        int N = iDims[1];

        // cuSolver(cuda 7.0) doesn't have support for M<N
        bool flip_and_transpose = M < N;

        if (flip_and_transpose) {
            std::swap(M, N);
            std::swap(vt, u);
        }

        int lwork = 0;
        CUSOLVER_CHECK(gesvd_buf_func<T>()(getDnHandle(), M, N, &lwork));
        T *lWorkspace = memAlloc<T>(lwork);
        //complex numbers would need rWorkspace
        //T *rWorkspace = memAlloc<T>(lwork);

        int *info = memAlloc<int>(1);

        if (flip_and_transpose) {
            transpose_inplace(in, true);
            CUSOLVER_CHECK(gesvd_func<T>()(getDnHandle(), 'A', 'A', M, N, in.get(),
                                           M, s.get(), u.get(), M, vt.get(), N,
                                           lWorkspace, lwork, NULL, info));
            std::swap(u, vt);
            transpose_inplace(vt, true);
        } else {
            Array<T> inCopy = copyArray<T>(in);
            CUSOLVER_CHECK(gesvd_func<T>()(getDnHandle(), 'A', 'A', M, N, in.get(),
                                           M, s.get(), u.get(), M, vt.get(), N,
                                           lWorkspace, lwork, NULL, info));
        }
        memFree(info);
        memFree(lWorkspace);
        //memFree(rWorkspace);
    }

    template <typename T>
    void svd(Array<T> &s, Array<T> &u, Array<T> &vt, const Array<T> &in)
    {
        Array<T> inCopy = copyArray<T>(in);
        svdInPlace(s, u, vt, inCopy);
    }

#define INSTANTIATE_SVD(T)                                                              \
    template void svd<T>(Array<T> &s, Array<T> &u, Array<T> &vt, const Array<T> &in);   \
    template void svdInPlace<T>(Array<T> &s, Array<T> &u, Array<T> &vt, Array<T> &in);   \

    INSTANTIATE_SVD(float)
    //INSTANTIATE_SVD(cfloat)
    INSTANTIATE_SVD(double)
    //INSTANTIATE_SVD(cdouble)
}

#else
namespace cuda
{
    template <typename T>
    void svd(Array<T> &s, Array<T> &u, Array<T> &vt, const Array<T> &in)
    {
        AF_ERROR("CUDA cusolver not available. Linear Algebra is disabled",
                 AF_ERR_NOT_CONFIGURED);
    }

#define INSTANTIATE_SVD(T)                                                              \
    template void svd<T>(Array<T> &s, Array<T> &u, Array<T> &vt, const Array<T> &in);   \

    INSTANTIATE_SVD(float)
    //INSTANTIATE_SVD(cfloat)
    INSTANTIATE_SVD(double)
    //INSTANTIATE_SVD(cdouble)
}
#endif
