
#include <hip/hip_runtime.h>
/*******************************************************
 * Copyright (c) 2014, ArrayFire
 * All rights reserved.
 *
 * This file is distributed under 3-clause BSD license.
 * The complete license agreement can be obtained at:
 * http://arrayfire.com/licenses/BSD-3-Clause
 ********************************************************/

// Parts of this code sourced from SnopyDogy
// https://gist.github.com/SnopyDogy/a9a22497a893ec86aa3e

#if defined(WITH_GRAPHICS)

#include <Array.hpp>
#include <image.hpp>
#include <err_cuda.hpp>
#include <debug_cuda.hpp>
#include <interopManager.hpp>

using af::dim4;

namespace cuda
{

template<typename T>
void copy_image(const Array<T> &in, const fg_image_handle image)
{
    InteropManager& intrpMngr = InteropManager::getInstance();

    cudaGraphicsResource *cudaPBOResource = intrpMngr.getBufferResource(image);

    const T *d_X = in.get();
    // Map resource. Copy data to PBO. Unmap resource.
    size_t num_bytes;
    T* d_pbo = NULL;
    cudaGraphicsMapResources(1, &cudaPBOResource, 0);
    cudaGraphicsResourceGetMappedPointer((void **)&d_pbo, &num_bytes, cudaPBOResource);
    cudaMemcpy(d_pbo, d_X, num_bytes, cudaMemcpyDeviceToDevice);
    cudaGraphicsUnmapResources(1, &cudaPBOResource, 0);

    // Unlock array
    // Not implemented yet
    // X.unlock();

    CheckGL("After cuda resource copy");

    POST_LAUNCH_CHECK();
}

#define INSTANTIATE(T)      \
    template void copy_image<T>(const Array<T> &in, const fg_image_handle image);

INSTANTIATE(float)
INSTANTIATE(double)
INSTANTIATE(int)
INSTANTIATE(uint)
INSTANTIATE(uchar)
INSTANTIATE(char)

}

#endif
