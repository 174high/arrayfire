
#include <hip/hip_runtime.h>
/*******************************************************
 * Copyright (c) 2014, ArrayFire
 * All rights reserved.
 *
 * This file is distributed under 3-clause BSD license.
 * The complete license agreement can be obtained at:
 * http://arrayfire.com/licenses/BSD-3-Clause
 ********************************************************/

// Parts of this code sourced from SnopyDogy
// https://gist.github.com/SnopyDogy/a9a22497a893ec86aa3e

#if defined(WITH_GRAPHICS)

#include <Array.hpp>
#include <image.hpp>
#include <err_cuda.hpp>
#include <debug_cuda.hpp>
#include <platform.hpp>
#include <graphics_common.hpp>

#include <cuda.h>
#include <cuda_runtime.h>
#include <cuda_gl_interop.h>

using af::dim4;

namespace cuda
{

typedef std::map<fg_image_handle, cudaGraphicsResource *> interop_t;
typedef interop_t::iterator iter_t;

// Manager Class for cudaPBOResource: calls garbage collection at the end of the program
class InteropManager
{
    private:
        interop_t interop_maps[DeviceManager::MAX_DEVICES];

    public:
        static InteropManager& getInstance();
        ~InteropManager();
        cudaGraphicsResource* getPBOResource(const fg_image_handle handle);

    protected:
        InteropManager() {}
        InteropManager(InteropManager const&);
        void operator=(InteropManager const&);
        void destroyResources();
};

void InteropManager::destroyResources()
{
    int n = getActiveDeviceId();
    for(iter_t iter = interop_maps[n].begin(); iter != interop_maps[n].end(); iter++)
        cudaGraphicsUnregisterResource(iter->second);
}

InteropManager::~InteropManager()
{
    for(int i = 0; i < getDeviceCount(); i++) {
        setDevice(i);
        destroyResources();
    }
}

InteropManager& InteropManager::getInstance()
{
    static InteropManager my_instance;
    return my_instance;
}

cudaGraphicsResource* InteropManager::getPBOResource(const fg_image_handle key)
{
    int device = getActiveDeviceId();

    if(interop_maps[device].find(key) == interop_maps[device].end()) {
        cudaGraphicsResource *cudaPBOResource;
        // Register PBO with CUDA
        cudaGraphicsGLRegisterBuffer(&cudaPBOResource, key->gl_PBO, cudaGraphicsMapFlagsWriteDiscard);
        interop_maps[device][key] = cudaPBOResource;
    }

    return interop_maps[device][key];
}

template<typename T>
void copy_image(const Array<T> &in, const fg_image_handle image)
{
    InteropManager& intrpMngr = InteropManager::getInstance();

    cudaGraphicsResource *cudaPBOResource = intrpMngr.getPBOResource(image);

    const T *d_X = in.get();
    // Map resource. Copy data to PBO. Unmap resource.
    size_t num_bytes;
    T* d_pbo = NULL;
    cudaGraphicsMapResources(1, &cudaPBOResource, 0);
    cudaGraphicsResourceGetMappedPointer((void **)&d_pbo, &num_bytes, cudaPBOResource);
    cudaMemcpy(d_pbo, d_X, num_bytes, cudaMemcpyDeviceToDevice);
    cudaGraphicsUnmapResources(1, &cudaPBOResource, 0);

    // Unlock array
    // Not implemented yet
    // X.unlock();

    CheckGL("After cuda resource copy");

    POST_LAUNCH_CHECK();
}

#define INSTANTIATE(T)      \
    template void copy_image<T>(const Array<T> &in, const fg_image_handle image);

INSTANTIATE(float)
INSTANTIATE(double)
INSTANTIATE(int)
INSTANTIATE(uint)
INSTANTIATE(uchar)
INSTANTIATE(char)

}

#endif
