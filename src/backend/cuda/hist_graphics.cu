
#include <hip/hip_runtime.h>
/*******************************************************
 * Copyright (c) 2014, ArrayFire
 * All rights reserved.
 *
 * This file is distributed under 3-clause BSD license.
 * The complete license agreement can be obtained at:
 * http://arrayfire.com/licenses/BSD-3-Clause
 ********************************************************/

#if defined (WITH_GRAPHICS)

#include <hist_graphics.hpp>
#include <err_cuda.hpp>

namespace cuda
{

template<typename T>
void copy_histogram(const Array<T> &data, const fg::Histogram* hist)
{
    CUDA_NOT_SUPPORTED();
}

#define INSTANTIATE(T)  \
    template void copy_histogram<T>(const Array<T> &data, const fg::Histogram* hist);

INSTANTIATE(float)
INSTANTIATE(int)
INSTANTIATE(uint)
INSTANTIATE(uchar)

}

#endif  // WITH_GRAPHICS
