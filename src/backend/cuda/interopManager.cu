
#include <hip/hip_runtime.h>
/*******************************************************
 * Copyright (c) 2014, ArrayFire
 * All rights reserved.
 *
 * This file is distributed under 3-clause BSD license.
 * The complete license agreement can be obtained at:
 * http://arrayfire.com/licenses/BSD-3-Clause
 ********************************************************/

// Parts of this code sourced from SnopyDogy
// https://gist.github.com/SnopyDogy/a9a22497a893ec86aa3e

#if defined(WITH_GRAPHICS)

#include <interopManager.hpp>

namespace cuda
{

void InteropManager::destroyResources()
{
    int n = getActiveDeviceId();
    for(iter_t iter = interop_maps[n].begin(); iter != interop_maps[n].end(); iter++)
        cudaGraphicsUnregisterResource(iter->second);
}

InteropManager::~InteropManager()
{
    for(int i = 0; i < getDeviceCount(); i++) {
        setDevice(i);
        destroyResources();
    }
}

InteropManager& InteropManager::getInstance()
{
    static InteropManager my_instance;
    return my_instance;
}

cudaGraphicsResource* InteropManager::getPBOResource(const fg_image_handle key)
{
    int device = getActiveDeviceId();

    if(interop_maps[device].find(key) == interop_maps[device].end()) {
        cudaGraphicsResource *cudaPBOResource;
        // Register PBO with CUDA
        cudaGraphicsGLRegisterBuffer(&cudaPBOResource, key->gl_PBO, cudaGraphicsMapFlagsWriteDiscard);
        interop_maps[device][key] = cudaPBOResource;
    }

    return interop_maps[device][key];
}

}

#endif
